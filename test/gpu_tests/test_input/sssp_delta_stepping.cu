#include "hip/hip_runtime.h"
#include "gpu_intrinsics.h"
#include <algorithm>

#define ITER_COUNT (1)
#define USE_DEDUP 0
#define SORT_NODES 0
#include <assert.h>
#include <vector>
#include <queue>

//#define DEBUG

typedef struct {
	int32_t *SP;
	int32_t *output_size;
	int32_t num_blocks;
	int32_t *node_borders;
	int32_t *edge_borders;
	int32_t *old_indices;
	int32_t window_lower;
	int32_t window_upper;		
	int32_t *new_window_start;
}algo_state;

int32_t __device__ *SP;
int32_t *__host_SP;
int32_t *__device_SP;

int32_t __device__ window_lower;
int32_t __device__ window_upper;


#define VIRTUAL_WARP_SIZE (32)
#define NUM_THREADS (1024)
#define NUM_BLOCKS (80)
#define CTA_SIZE (1024)
#define WARP_SIZE (32)
#define STAGE_1_SIZE (8)

void __global__ init_kernel(gpu_runtime::GraphT<int32_t> graph, algo_state device_state, int start_v) {
        int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x * gridDim.x;
        int total_work = graph.num_vertices;
        int work_per_thread = (total_work + num_threads - 1)/num_threads;
        for (int i = 0; i < work_per_thread; i++) {
                int id = num_threads * i + thread_id;
                if (id < total_work) {
					device_state.SP[id] = INT_MAX;
                }
        }
	if (thread_id == 0) {
		//reset with the new data structure
		SP[start_v] = 0;
		device_state.SP[start_v] = 0;
	}
}

bool __device__ updateEdge(int32_t src, int32_t dst, int32_t weight) {
	bool output2;
	bool SP_trackving_var_1 = 0;
	SP_trackving_var_1 = gpu_runtime::writeMin(&SP[dst], (SP[src] + weight));
	output2 = SP_trackving_var_1;

	if (SP[dst] >= window_upper) return false;
	
	return output2;
}

template <typename EdgeWeightType>
void __device__ gpu_operator_body_3(gpu_runtime::GraphT<EdgeWeightType> graph, int32_t src, int32_t dst, int32_t edge_id, gpu_runtime::VertexFrontier input_frontier, gpu_runtime::VertexFrontier output_frontier) {
	// Body of the actual operator code
	EdgeWeightType weight = graph.d_edge_weight[edge_id];
	if (updateEdge(src, dst, weight)){
		//gpu_runtime::enqueueVertexSparseQueue(output_frontier.d_sparse_queue_output, output_frontier.d_num_elems_output, dst);
		gpu_runtime::enqueueVertexBytemap(output_frontier.d_byte_map_output, output_frontier.d_num_elems_output, dst);
	}
}

void __global__ update_nodes_identify_min(gpu_runtime::GraphT<int32_t> graph, algo_state device_state) {
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;
	
	int total_work = graph.num_vertices;
	int work_per_thread = (total_work + num_threads - 1)/num_threads;
	int32_t my_minimum = INT_MAX;
	for (int i = 0; i < work_per_thread; i++) {
		int32_t node_id = thread_id + i * num_threads;
		if (node_id < graph.num_vertices) {
			if (SP[node_id] >= device_state.window_upper && SP[node_id] != INT_MAX && SP[node_id] < my_minimum) {
				my_minimum = SP[node_id];
			}
		}
	}
	if (my_minimum < device_state.new_window_start[0]) {
		atomicMin(device_state.new_window_start, my_minimum);
	}	
}
void __global__ update_nodes_special(gpu_runtime::GraphT<int32_t> graph, algo_state device_state,  gpu_runtime::VertexFrontier output_frontier) {
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;
	//int warp_id = thread_id / 32;	
	
	int total_work = graph.num_vertices;
	int work_per_thread = (total_work + num_threads - 1)/num_threads;
	for (int i = 0; i < work_per_thread; i++) {
		int32_t node_id = thread_id + i * num_threads;
		if (node_id < graph.num_vertices) {
			if(SP[node_id] >= device_state.window_lower && SP[node_id] < device_state.window_upper) {
				gpu_runtime::enqueueVertexSparseQueue(output_frontier.d_sparse_queue_output, output_frontier.d_num_elems_output, node_id);
			}	
		}
	}
}
void allocate_state(algo_state &host_state, algo_state &device_state, gpu_runtime::GraphT<int32_t> &graph) {
	host_state.SP = new int[graph.num_vertices];
	host_state.output_size = new int32_t[1];
	host_state.new_window_start = new int32_t[1];
	hipMalloc(&device_state.SP, sizeof(int32_t)*graph.num_vertices);	
	hipMalloc(&device_state.output_size, sizeof(int32_t));
	hipMalloc(&device_state.new_window_start, sizeof(int32_t));
}

void swap_pointers(int32_t **a, int32_t **b) {
	int32_t* t = *a;
	*a = *b;
	*b = t;
}

void __device__ SP_generated_vector_op_apply_func_0(int32_t v) {
	SP[v] = 2147483647;
}


int main(int argc, char *argv[]) {
	hipSetDevice(0);
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	gpu_runtime::GraphT<int32_t> graph;
	gpu_runtime::load_graph(graph, argv[1], false);
	int32_t delta = atoi(argv[3]);
	int32_t start_vertex = atoi(argv[2]);
	
	hipMalloc(&__device_SP, gpu_runtime::builtin_getVertices(graph) * sizeof(int32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(SP), &__device_SP, sizeof(int32_t*), 0);
	__host_SP = new int32_t[gpu_runtime::builtin_getVertices(graph)];
	gpu_runtime::vertex_set_apply_kernel<gpu_runtime::AccessorAll, SP_generated_vector_op_apply_func_0><<<NUM_CTA, CTA_SIZE>>>(graph.getFullFrontier());
	
	algo_state host_state, device_state;	
	allocate_state(host_state, device_state, graph);
	
	host_state.window_lower = 0;
	host_state.window_upper = delta;
	device_state.window_lower = 0;
	device_state.window_upper = delta;

   


	hipDeviceSynchronize();
	
	float total_time = 0;
	for (int outer = 0; outer < ITER_COUNT; outer++) {
		float iter_total = 0;
		//this sets it to Sparse
		gpu_runtime::VertexFrontier frontier = gpu_runtime::create_new_vertex_set(gpu_runtime::builtin_getVertices(graph));
		gpu_runtime::builtin_addVertex(frontier, start_vertex);
		startTimer();
		
		init_kernel<<<NUM_BLOCKS, CTA_SIZE>>>(graph, device_state, start_vertex);
		gpu_runtime::cudaCheckLastError();
		
		int iters = 0;	
		hipDeviceSynchronize();
		float t = stopTimer();
		//printf("Init time = %f\n", t);
		iter_total+=t;

		while(gpu_runtime::builtin_getVertexSetSize(frontier) != (0)){
			startTimer();
			iters++;
			gpu_runtime::vertex_set_prepare_sparse(frontier);
			hipMemcpyToSymbol(HIP_SYMBOL(window_upper), &device_state.window_upper, sizeof(int32_t), 0);
			gpu_runtime::cudaCheckLastError();
			gpu_runtime::vertex_based_load_balance_host<int32_t, gpu_operator_body_3, gpu_runtime::AccessorSparse, gpu_runtime::true_function>(graph, frontier, frontier);  
			
			gpu_runtime::swap_bytemaps(frontier);
			// set the input to the prepare function
			frontier.format_ready = gpu_runtime::VertexFrontier::BYTEMAP;
			
			if (gpu_runtime::builtin_getVertexSetSize(frontier) == (0)) {
				host_state.new_window_start[0] = INT_MAX;
				hipMemcpy(device_state.new_window_start, host_state.new_window_start, sizeof(int32_t), hipMemcpyHostToDevice);

				//should not need to change 
				update_nodes_identify_min<<<NUM_BLOCKS, CTA_SIZE>>>(graph, device_state);	
				hipMemcpy(host_state.new_window_start, device_state.new_window_start, sizeof(int32_t), hipMemcpyDeviceToHost);

				//this is for termination when it is all finished
				if (host_state.new_window_start[0] == INT_MAX) {
					break;
				}

				//if it is not a pointer, then you can set by value directly
				device_state.window_lower = host_state.new_window_start[0];
				device_state.window_upper = host_state.new_window_start[0] + delta;

				update_nodes_special<<<NUM_BLOCKS, CTA_SIZE>>>( graph, device_state, frontier);
				gpu_runtime::swap_queues(frontier);
				frontier.format_ready = gpu_runtime::VertexFrontier::SPARSE; 
				
			}

			t = stopTimer();

			#ifdef DEBUG
			//printf("Iter %d output_size = %d \n", iters, gpu_runtime::builtin_getVertexSetSize(frontier));
			#endif
			
			iter_total += t;
		}


		#ifdef DEBUG
		printf("Num iters = %d\n", iters);
		#endif
		//printf("Time elapsed = %f\n", iter_total);
		total_time += iter_total;

	}
	//printf("Total time = %f\n", total_time);
	if (argc > 3)
		if (argv[4][0] == 'v'){ 
			//FILE *output = fopen("output.txt", "w");
			hipMemcpy(host_state.SP, __device_SP, sizeof(int32_t)*graph.num_vertices, hipMemcpyDeviceToHost);
			for (int i = 0; i < graph.num_vertices; i++)
				#ifdef DEBUG
				printf("%d, %d\n", i, host_state.SP[i]);
				#else
				printf("%d\n", host_state.SP[i]);
                #endif
		}
	return 0;
}
