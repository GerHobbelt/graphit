#include "hip/hip_runtime.h"
#include "gpu_intrinsics.h"
#include <algorithm>


#define USE_DEDUP 0
#define SORT_NODES 0
#include <assert.h>
#include <vector>
#include <queue>

//#define DEBUG

#ifdef DEBUG
  #define ITER_COUNT (5)
#else
  #define ITER_COUNT (1)
#endif

gpu_runtime::GPUPriorityQueue<int> host_gpq;
gpu_runtime::GPUPriorityQueue<int> __device__  device_gpq; 

typedef struct {
	int32_t *SP;
	int32_t *output_size;
	int32_t num_blocks;
	int32_t *node_borders;
	int32_t *edge_borders;
	int32_t *old_indices;
	int32_t window_lower;
	int32_t window_upper;		
	int32_t *new_window_start;
}algo_state;

int32_t __device__ *SP;
int32_t *__host_SP;
int32_t *__device_SP;

//int32_t __device__ window_lower;
//int32_t __device__ window_upper;


#define VIRTUAL_WARP_SIZE (32)
#define NUM_THREADS (1024)
#define NUM_BLOCKS (80)
#define CTA_SIZE (1024)
#define WARP_SIZE (32)
#define STAGE_1_SIZE (8)

void __global__ init_kernel(gpu_runtime::GraphT<int32_t> graph, algo_state device_state, int start_v) {
        int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
        int num_threads = blockDim.x * gridDim.x;
        int total_work = graph.num_vertices;
        int work_per_thread = (total_work + num_threads - 1)/num_threads;
        for (int i = 0; i < work_per_thread; i++) {
                int id = num_threads * i + thread_id;
                if (id < total_work) {
					device_state.SP[id] = INT_MAX;
                }
        }
	if (thread_id == 0) {
		//reset with the new data structure
		SP[start_v] = 0;
		device_state.SP[start_v] = 0;
	}
}

bool __device__ updateEdge(int32_t src, int32_t dst, int32_t weight) {
	bool output2;
	bool SP_trackving_var_1 = 0;
	SP_trackving_var_1 = gpu_runtime::writeMin(&SP[dst], (SP[src] + weight));
	output2 = SP_trackving_var_1;

	//if (SP[dst] >= window_upper) return false;
	if (SP[dst] >= (device_gpq.current_priority_ + device_gpq.delta_)) return false;
	
	return output2;
}

template <typename EdgeWeightType>
void __device__ gpu_operator_body_3(gpu_runtime::GraphT<EdgeWeightType> graph, int32_t src, int32_t dst, int32_t edge_id, gpu_runtime::VertexFrontier input_frontier, gpu_runtime::VertexFrontier output_frontier) {
	// Body of the actual operator code
	EdgeWeightType weight = graph.d_edge_weight[edge_id];
	if (updateEdge(src, dst, weight)){
		//gpu_runtime::enqueueVertexSparseQueue(output_frontier.d_sparse_queue_output, output_frontier.d_num_elems_output, dst);
		gpu_runtime::enqueueVertexBytemap(output_frontier.d_byte_map_output, output_frontier.d_num_elems_output, dst);
	}
}

void __global__ update_nodes_identify_min(gpu_runtime::GraphT<int32_t> graph, algo_state device_state) {
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;
	
	int total_work = graph.num_vertices;
	int work_per_thread = (total_work + num_threads - 1)/num_threads;
	int32_t my_minimum = INT_MAX;
	for (int i = 0; i < work_per_thread; i++) {
		int32_t node_id = thread_id + i * num_threads;
		if (node_id < graph.num_vertices) {
		  if (SP[node_id] >= (device_gpq.window_upper_) && SP[node_id] != INT_MAX && SP[node_id] < my_minimum) {
				my_minimum = SP[node_id];
			}
		}
	}
	//if (my_minimum < device_state.new_window_start[0]) {
	if (my_minimum < device_gpq.current_priority_){
	  //atomicMin(device_state.new_window_start, my_minimum);
	  atomicMin(&(device_gpq.current_priority_), my_minimum);
	}	
}
void __global__ update_nodes_special(gpu_runtime::GraphT<int32_t> graph, algo_state device_state,  gpu_runtime::VertexFrontier output_frontier) {
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int num_threads = blockDim.x * gridDim.x;
	//int warp_id = thread_id / 32;	
	
	int total_work = graph.num_vertices;
	int work_per_thread = (total_work + num_threads - 1)/num_threads;
	for (int i = 0; i < work_per_thread; i++) {
		int32_t node_id = thread_id + i * num_threads;
		if (node_id < graph.num_vertices) {
		  //if(SP[node_id] >= device_state.window_lower && SP[node_id] < device_state.window_upper) {
		  if(SP[node_id] >= device_gpq.current_priority_ && SP[node_id] < (device_gpq.current_priority_ + device_gpq.delta_)) {
				gpu_runtime::enqueueVertexSparseQueue(output_frontier.d_sparse_queue_output, output_frontier.d_num_elems_output, node_id);
			}	
		}
	}
}
void allocate_state(algo_state &host_state, algo_state &device_state, gpu_runtime::GraphT<int32_t> &graph) {
	host_state.SP = new int[graph.num_vertices];
	host_state.output_size = new int32_t[1];
	host_state.new_window_start = new int32_t[1];
	hipMalloc(&device_state.SP, sizeof(int32_t)*graph.num_vertices);	
	hipMalloc(&device_state.output_size, sizeof(int32_t));
	hipMalloc(&device_state.new_window_start, sizeof(int32_t));
}

void swap_pointers(int32_t **a, int32_t **b) {
	int32_t* t = *a;
	*a = *b;
	*b = t;
}

void __device__ SP_generated_vector_op_apply_func_0(int32_t v) {
	SP[v] = 2147483647;
}


int main(int argc, char *argv[]) {
	hipSetDevice(0);
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	gpu_runtime::GraphT<int32_t> graph;
	gpu_runtime::load_graph(graph, argv[1], false);
	int32_t delta = atoi(argv[3]);
	int32_t start_vertex = atoi(argv[2]);
	
	hipMalloc(&__device_SP, gpu_runtime::builtin_getVertices(graph) * sizeof(int32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(SP), &__device_SP, sizeof(int32_t*), 0);
	
	__host_SP = new int32_t[gpu_runtime::builtin_getVertices(graph)];

	algo_state host_state, device_state;	
	allocate_state(host_state, device_state, graph);

	hipDeviceSynchronize();
	
	float total_time = 0;
	for (int outer = 0; outer < ITER_COUNT; outer++) {
		float iter_total = 0;
		//this sets it to Sparse
		gpu_runtime::VertexFrontier frontier = gpu_runtime::create_new_vertex_set(gpu_runtime::builtin_getVertices(graph));
		gpu_runtime::builtin_addVertex(frontier, start_vertex);
		gpu_runtime::vertex_set_apply_kernel<gpu_runtime::AccessorAll, SP_generated_vector_op_apply_func_0><<<NUM_CTA, CTA_SIZE>>>(graph.getFullFrontier());
		startTimer();

		host_gpq.delta_ = delta;
		host_gpq.current_priority_ = 0 ;

		hipMemcpyToSymbol(HIP_SYMBOL(device_gpq), &host_gpq, sizeof(host_gpq), 0);
		gpu_runtime::cudaCheckLastError();
		
		init_kernel<<<NUM_BLOCKS, CTA_SIZE>>>(graph, device_state, start_vertex);
		gpu_runtime::cudaCheckLastError();
		
		int iters = 0;	
		hipDeviceSynchronize();
		float t = stopTimer();
		//printf("Init time = %f\n", t);
		iter_total+=t;

		//while(gpu_runtime::builtin_getVertexSetSize(frontier) != (0)){
		while(! host_gpq.finished()){
			startTimer();
			iters++;
			gpu_runtime::vertex_set_prepare_sparse(frontier);
			//hipMemcpyToSymbol(HIP_SYMBOL(window_upper), &device_state.window_upper, sizeof(int32_t), 0);
			//Might not be necessary, always synchronized at this point?? 
			hipMemcpyToSymbol(HIP_SYMBOL(device_gpq), &host_gpq, sizeof(host_gpq), 0);
			gpu_runtime::cudaCheckLastError();

			//gpu_runtime::vertex_based_load_balance_host<int32_t, gpu_operator_body_3, gpu_runtime::AccessorSparse, gpu_runtime::true_function>(graph, frontier, frontier);  
			gpu_runtime::TWCE_load_balance_host<int32_t, gpu_operator_body_3, gpu_runtime::AccessorSparse, gpu_runtime::true_function>(graph, frontier, frontier);
			gpu_runtime::cudaCheckLastError();
			
			gpu_runtime::swap_bytemaps(frontier);
			// set the input to the prepare function
			frontier.format_ready = gpu_runtime::VertexFrontier::BYTEMAP;
			
			if (gpu_runtime::builtin_getVertexSetSize(frontier) == (0)) {
			  //host_state.new_window_start[0] = INT_MAX;
			  host_gpq.window_upper_ = host_gpq.current_priority_ + host_gpq.delta_;
			  host_gpq.current_priority_ = INT_MAX;
			  
			  hipMemcpyToSymbol(HIP_SYMBOL(device_gpq), &host_gpq, sizeof(host_gpq), 0);
			  gpu_runtime::cudaCheckLastError();

			  update_nodes_identify_min<<<NUM_BLOCKS, CTA_SIZE>>>(graph, device_state);
			  gpu_runtime::cudaCheckLastError();
			  hipMemcpyFromSymbol(&host_gpq, HIP_SYMBOL(device_gpq), sizeof(host_gpq), 0,hipMemcpyDeviceToHost);
			  gpu_runtime::cudaCheckLastError();

			  //if(host_gpq.current_priority_ == INT_MAX){
			  //  break;
			  //}			  
			  update_nodes_special<<<NUM_BLOCKS, CTA_SIZE>>>( graph, device_state, frontier);
			  gpu_runtime::cudaCheckLastError();
			  gpu_runtime::swap_queues(frontier);
			  frontier.format_ready = gpu_runtime::VertexFrontier::SPARSE; 
			}

			hipDeviceSynchronize();
			t = stopTimer();

			#ifdef DEBUG
			//printf("Iter %d output_size = %d \n", iters, gpu_runtime::builtin_getVertexSetSize(frontier));
			#endif
			
			iter_total += t;
		}


		#ifdef DEBUG
		printf("Num iters = %d\n", iters);
		printf("Time elapsed = %f\n", iter_total);
		#endif
		
		total_time += iter_total;

	}

	#ifdef DEBUG
	printf("Total time = %f\n", total_time);
	#endif
	
	if (argc > 3)
		if (argv[4][0] == 'v'){ 
			//FILE *output = fopen("output.txt", "w");
			hipMemcpy(host_state.SP, __device_SP, sizeof(int32_t)*graph.num_vertices, hipMemcpyDeviceToHost);
			#ifdef DEBUG
			FILE *output = fopen("output.txt", "w");
			#endif
			
			for (int i = 0; i < graph.num_vertices; i++){
				#ifdef DEBUG
				fprintf(output, "%d, %d\n", i, host_state.SP[i]);
				#else
				printf("%d\n", host_state.SP[i]);
                                #endif
			}
		}
	return 0;
}
